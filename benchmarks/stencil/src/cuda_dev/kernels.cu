#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include "common.h"


__global__ void block2D_reg_tiling(float c0,float c1,float *current,float *next, int nx, int ny, int nz)
{
    extern __shared__ float shared[];
    int size=blockDim.x*blockDim.y;
    int tx=blockDim.x;
    float * A1=&shared[0];
    float * A2=&shared[size];
    float * A3=&shared[2*size];
    float * Anew=&shared[3*size];
    float * temp;

    int i = blockIdx.x*blockDim.x+threadIdx.x;
    int j = blockIdx.y*blockDim.y+threadIdx.y;

    int id=threadIdx.x+tx*threadIdx.y;
    A1[id]=current[Index3D(nx, ny, i, j, 0)] ;
    A2[id]=current[Index3D(nx, ny, i, j, 1)] ;
    __syncthreads();
    for(int k=1;k<nz-1;k++)
    {
        A3[id]=current[Index3D (nx, ny, i, j, k+1)] ;
        if( i>0 && j>0 &&(i<nx-1) &&(j<ny-1) )
        {
            __syncthreads();
            Anew[id]=0;
            if(threadIdx.x==0)
                Anew[id]+=current[Index3D (nx, ny, i - 1, j, k)];
            else
                Anew[id]+=A2[id-1];//current[Index3D (nx, ny, i - 1, j, k)];

            if((threadIdx.x==blockDim.x-1)||(i==nx-2))
                Anew[id]+=current[Index3D (nx, ny, i + 1, j, k)];
            else
                Anew[id]+=A2[id+1];//current[Index3D (nx, ny, i + 1, j, k)];

            if(threadIdx.y==0)
                Anew[id]+=current[Index3D (nx, ny, i , j - 1, k)];
            else
                Anew[id]+=A2[id-tx];//current[Index3D (nx, ny, i , j - 1, k)];

            if((threadIdx.y==blockDim.y-1)||(j==ny-2))
                Anew[id]+=current[Index3D (nx, ny, i , j + 1, k)];
            else
                Anew[id]+=A2[id+tx];//current[Index3D (nx, ny, i , j + 1, k)];

            next[Index3D (nx, ny, i, j, k)]=c1*(A1[id] + A3[id] + Anew[id])-(c0*A2[id]);

        }
        temp=A1;
        A1=A2;
        A2=A3;
        A3=temp;
    }

}


